#include "hip/hip_runtime.h"
//
// Created by Bruno on 26/11/2018.
//

#include "matrixCuda.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void mult(float *res, float *m1, int rows_m1, int cols_m1, float *m2, int rows_m2, int cols_m2){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = y * cols_m2 + x;

    if(tid < rows_m1 * cols_m2){
        res[tid] = 0;
        for(int i = 0; i < cols_m1; i++){
            res[tid] += m1[y*cols_m1 + i] * m2[i*cols_m2 + x];
        }
    }
}

float *transfer_matrix2device(float *m1, float *d_m1, int rows, int cols){
    hipMemcpy(d_m1, m1, sizeof(float) * rows * cols, hipMemcpyHostToDevice);
}
float *transfer_matrix2host(float *m1, float *d_m1, int rows, int cols){
    hipMemcpy(m1, d_m1, sizeof(float) * rows * cols, hipMemcpyDeviceToHost);
}

int matrix_mult(float *result, float *m1, int rows_m1, int cols_m1, float *m2, int rows_m2, int cols_m2){
    if(cols_m1 != rows_m2) return 0;

    dim3 blockDim(rows_m1 / 2 + rows_m1 % 2 > 0, cols_m2/2 + cols_m2 % 2 > 0);
    dim3 gridDim(rows_m1 / 2, cols_m2/2);

    mult<<<gridDim, blockDim>>>(result, m1, rows_m1, cols_m1, m2, rows_m2, cols_m2);

}

float *allocate_matrix(int rows, int cols){
    float *matrix;
    hipMalloc(&matrix, sizeof(float) * rows * cols);

    return matrix;
}
